#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#define N 5
#define  INF 9999

int matrixGraph[N][N] = {
	{ 0, 5, INF, 3, INF },
	{ INF, 0, 6, INF, 4 },
	{ INF, INF, 0, 9, INF },
	{ INF, INF, INF, 0, INF },
	{ INF, INF, INF, 2, 0 }
};

__global__ void RoyFloyd (int matrixGraph[N][N])
{
	
	int i = threadIdx.x;
	int j = threadIdx.y;
	for (int k = 0; k < N; k++)
	{
		if (matrixGraph[i][k] + matrixGraph[k][j] < matrixGraph[i][j]) {
			matrixGraph[i][j] = matrixGraph[i][k] + matrixGraph[k][j];
		}
	}
}

int main()
{
	int *matrix;

	hipMalloc(&matrix, N*N * sizeof(int));
	for (int k = 0; k < N; ++k)
	{
		hipMemcpy(matrix, matrixGraph, N * N * sizeof(int), hipMemcpyHostToDevice);
		int* d_k;
		hipMalloc(&d_k, sizeof(int));
		hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);
		int numBlocks = 1;
		dim3 threadsPerBlock(N, N);
		RoyFloyd << < numBlocks, threadsPerBlock >> > (matrix);

		hipMemcpy(matrixGraph, matrix, N * N * sizeof(int), hipMemcpyDeviceToHost);
	}
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j)
		{
			if (matrixGraph[i][j] == INF)
				std::cout << "inf ,";
			else
				std::cout << matrixGraph[i][j] << ", ";
		}
		std::cout << std::endl;
	}

    hipFree(matrixGraph);
    hipFree(matrix);
	system("pause");
    return 0;

}
